#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/host_vector.h>
#include <thrust/complex.h>
#include <thrust/transform.h>

struct basic_arithmetic_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x,
				const thrust::complex<T> &y)
  {
    // exercise unary and binary arithmetic operators
    // Should return approximately 1
    return (+x + +y) + (x * y) / (y * x) + (-y + -x);
  } // end operator()()
}; // end make_pair_functor

struct general_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    // exercise general functions
    // Should return approximately 1
    return thrust::proj( (thrust::polar(abs(x),arg(x)) * conj(x))/norm(x));
  } // end operator()()
}; // end make_pair_functor

struct power_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x,
				const thrust::complex<T> &y)
  {
    // exercise power functions
    return pow(x,y)+sqrt(x);
  } // end operator()()
}; // end make_pair_functor

struct exponential_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    // exercise power functions
    // should return approximately 1
    return log(exp(x))/(T(2.30258509299404568402)*log10(exp(x)));
  } // end operator()()
}; // end make_pair_functor

struct trigonometric_functor
{
  template<typename T>
  __host__ __device__
  thrust::complex<T> operator()(const thrust::complex<T> &x)
  {
    // exercise power functions
    // might return approximately 1
    return acos(cos(x))+asin(sin(x))-T(4.0)*x
      +(acosh(cosh(x)) + asinh(sinh(x)));// + atanh(tanh(x)));
//+atan(tan(x));
      //      (acosh(cosh(x)) + asinh(sinh(x)) + atanh(tanh(x)));
  } // end operator()()
}; // end make_pair_functor


template <typename T>
struct TestComplexTransform
{
  void operator()(const size_t n)
  {
    typedef thrust::complex<T> type;

    thrust::host_vector<T> real = unittest::random_samples<T>(n);
    thrust::host_vector<T> imag = unittest::random_samples<T>(n);
    thrust::host_vector<type> h_p1(n);
    thrust::host_vector<type> h_p2(n);

    for(size_t i = 0; i<n; i++){
      h_p1[i].real(real[i]);
      h_p1[i].imag(imag[i]);
    }

    real = unittest::random_samples<T>(n);
    imag = unittest::random_samples<T>(n);
    for(size_t i = 0; i<n; i++){
      h_p2[i].real(real[i]);
      h_p2[i].imag(imag[i]);
    }
    thrust::host_vector<type>   h_result(n);

    thrust::device_vector<type> d_p1 = h_p1;
    thrust::device_vector<type> d_p2 = h_p2;
    thrust::device_vector<type> d_result(n);

    // run basic arithmetic on the host
    thrust::transform(h_p1.begin(), h_p1.end(), h_p2.begin(), h_result.begin(), basic_arithmetic_functor());
    // run basic arithmetic on the host
    thrust::transform(d_p1.begin(), d_p1.end(), d_p2.begin(), d_result.begin(), basic_arithmetic_functor());    
    // Currently just checking for compilation
    ASSERT_ALMOST_EQUAL(h_result, d_result);
    
    // run general functions on the host
    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), general_functor());
    // run general functions on the host
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), general_functor());
    // Currently just checking for compilation
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    // run power functions on the host
    thrust::transform(h_p1.begin(), h_p1.end(), h_p2.begin(), h_result.begin(), power_functor());
    // run power functions on the host
    thrust::transform(d_p1.begin(), d_p1.end(), d_p2.begin(), d_result.begin(), power_functor());
    // Currently just checking for compilation
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    // run exponential functions on the host
    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), exponential_functor());
    // run exponential functions on the host
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), exponential_functor());
    // Currently just checking for compilation
    ASSERT_ALMOST_EQUAL(h_result, d_result);

    // run trigonometric functions on the host
    thrust::transform(h_p1.begin(), h_p1.end(), h_result.begin(), trigonometric_functor());
    // run trigonometric functions on the host
    thrust::transform(d_p1.begin(), d_p1.end(), d_result.begin(), trigonometric_functor());
    // Currently just checking for compilation
    ASSERT_ALMOST_EQUAL(h_result, d_result);
  }
};
VariableUnitTest<TestComplexTransform, FloatingPointTypes> TestComplexTransformInstance;
