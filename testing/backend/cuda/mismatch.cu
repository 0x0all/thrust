#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/mismatch.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename Iterator3>
__global__ void mismatch_kernel(Iterator1 first1, Iterator1 last1, Iterator2 first2, Iterator3 result)
{
  *result = thrust::mismatch(thrust::seq, first1, last1, first2);
}


void TestMismatchDeviceSeq()
{
  thrust::device_vector<int> a(4);
  thrust::device_vector<int> b(4);
  a[0] = 1; b[0] = 1;
  a[1] = 2; b[1] = 2;
  a[2] = 3; b[2] = 4;
  a[3] = 4; b[3] = 3;

  typedef thrust::pair<
    typename thrust::device_vector<int>::iterator,
    typename thrust::device_vector<int>::iterator
  > pair_type;

  thrust::device_vector<pair_type> d_result(1);
  
  mismatch_kernel<<<1,1>>>(a.begin(), a.end(), b.begin(), d_result.begin());

  ASSERT_EQUAL(2, ((pair_type)d_result[0]).first  - a.begin());
  ASSERT_EQUAL(2, ((pair_type)d_result[0]).second - b.begin());
  
  b[2] = 3;
  
  mismatch_kernel<<<1,1>>>(a.begin(), a.end(), b.begin(), d_result.begin());
  ASSERT_EQUAL(3, ((pair_type)d_result[0]).first  - a.begin());
  ASSERT_EQUAL(3, ((pair_type)d_result[0]).second - b.begin());
  
  b[3] = 4;
  
  mismatch_kernel<<<1,1>>>(a.begin(), a.end(), b.begin(), d_result.begin());
  ASSERT_EQUAL(4, ((pair_type)d_result[0]).first  - a.begin());
  ASSERT_EQUAL(4, ((pair_type)d_result[0]).second - b.begin());
}
DECLARE_UNITTEST(TestMismatchDeviceSeq);


void TestMismatchCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;

  Vector a(4); Vector b(4);
  a[0] = 1; b[0] = 1;
  a[1] = 2; b[1] = 2;
  a[2] = 3; b[2] = 4;
  a[3] = 4; b[3] = 3;

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL(thrust::mismatch(thrust::cuda::par(s), a.begin(), a.end(), b.begin()).first  - a.begin(), 2);
  ASSERT_EQUAL(thrust::mismatch(thrust::cuda::par(s), a.begin(), a.end(), b.begin()).second - b.begin(), 2);

  b[2] = 3;
  
  ASSERT_EQUAL(thrust::mismatch(thrust::cuda::par(s), a.begin(), a.end(), b.begin()).first  - a.begin(), 3);
  ASSERT_EQUAL(thrust::mismatch(thrust::cuda::par(s), a.begin(), a.end(), b.begin()).second - b.begin(), 3);
  
  b[3] = 4;
  
  ASSERT_EQUAL(thrust::mismatch(thrust::cuda::par(s), a.begin(), a.end(), b.begin()).first  - a.begin(), 4);
  ASSERT_EQUAL(thrust::mismatch(thrust::cuda::par(s), a.begin(), a.end(), b.begin()).second - b.begin(), 4);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMismatchCudaStreams);

