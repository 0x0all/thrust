#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/merge.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


template<typename Iterator1,
         typename Iterator2,
         typename Iterator3,
         typename Iterator4,
         typename Iterator5,
         typename Iterator6,
         typename Iterator7>
__global__
void merge_by_key_kernel(Iterator1 keys_first1, Iterator1 keys_last1,
                         Iterator2 keys_first2, Iterator2 keys_last2,
                         Iterator3 values_first1,
                         Iterator4 values_first2,
                         Iterator5 keys_result,
                         Iterator6 values_result,
                         Iterator7 result)
{
  *result = thrust::merge_by_key(thrust::seq, keys_first1, keys_last1, keys_first2, keys_last2, values_first1, values_first2, keys_result, values_result);
}


template<typename T>
void TestMergeByKeyDeviceSeq(size_t n)
{
  thrust::host_vector<T> random_keys = unittest::random_integers<unittest::int8_t>(n);
  thrust::host_vector<T> random_vals = unittest::random_integers<unittest::int8_t>(n);

  size_t denominators[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
  size_t num_denominators = sizeof(denominators) / sizeof(size_t);

  for(size_t i = 0; i < num_denominators; ++i)
  {
    size_t size_a = n / denominators[i];

    thrust::host_vector<T> h_a_keys(random_keys.begin(), random_keys.begin() + size_a);
    thrust::host_vector<T> h_b_keys(random_keys.begin() + size_a, random_keys.end());

    thrust::host_vector<T> h_a_vals(random_vals.begin(), random_vals.begin() + size_a);
    thrust::host_vector<T> h_b_vals(random_vals.begin() + size_a, random_vals.end());

    thrust::stable_sort(h_a_keys.begin(), h_a_keys.end());
    thrust::stable_sort(h_b_keys.begin(), h_b_keys.end());

    thrust::device_vector<T> d_a_keys = h_a_keys;
    thrust::device_vector<T> d_b_keys = h_b_keys;

    thrust::device_vector<T> d_a_vals = h_a_vals;
    thrust::device_vector<T> d_b_vals = h_b_vals;

    thrust::host_vector<T> h_result_keys(n);
    thrust::host_vector<T> h_result_vals(n);

    thrust::device_vector<T> d_result_keys(n);
    thrust::device_vector<T> d_result_vals(n);

    thrust::pair<
      typename thrust::host_vector<T>::iterator,
      typename thrust::host_vector<T>::iterator
    > h_end;

    typedef thrust::pair<
      typename thrust::device_vector<T>::iterator,
      typename thrust::device_vector<T>::iterator
    > iter_pair_type;

    thrust::device_vector<iter_pair_type> d_end_vec(1);

    h_end = thrust::merge_by_key(h_a_keys.begin(), h_a_keys.end(),
                                 h_b_keys.begin(), h_b_keys.end(),
                                 h_a_vals.begin(),
                                 h_b_vals.begin(),
                                 h_result_keys.begin(),
                                 h_result_vals.begin());
    h_result_keys.erase(h_end.first, h_result_keys.end());
    h_result_vals.erase(h_end.second, h_result_vals.end());

    merge_by_key_kernel<<<1,1>>>(d_a_keys.begin(), d_a_keys.end(),
                                 d_b_keys.begin(), d_b_keys.end(),
                                 d_a_vals.begin(),
                                 d_b_vals.begin(),
                                 d_result_keys.begin(),
                                 d_result_vals.begin(),
                                 d_end_vec.begin());

    iter_pair_type d_end = d_end_vec[0];

    d_result_keys.erase(d_end.first, d_result_keys.end());
    d_result_vals.erase(d_end.second, d_result_vals.end());

    ASSERT_EQUAL(h_result_keys, d_result_keys);
    ASSERT_EQUAL(h_result_vals, d_result_vals);
  }
}
DECLARE_VARIABLE_UNITTEST(TestMergeByKeyDeviceSeq);


void TestMergeByKeyCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::iterator Iterator;

  Vector a_key(3), a_val(3), b_key(4), b_val(4);

  a_key[0] = 0;  a_key[1] = 2; a_key[2] = 4;
  a_val[0] = 13; a_val[1] = 7; a_val[2] = 42;

  b_key[0] = 0 ; b_key[1] = 3;  b_key[2] = 3; b_key[3] = 4;
  b_val[0] = 42; b_val[1] = 42; b_val[2] = 7; b_val[3] = 13;

  Vector ref_key(7), ref_val(7);
  ref_key[0] = 0; ref_val[0] = 13;
  ref_key[1] = 0; ref_val[1] = 42;
  ref_key[2] = 2; ref_val[2] = 7;
  ref_key[3] = 3; ref_val[3] = 42;
  ref_key[4] = 3; ref_val[4] = 7;
  ref_key[5] = 4; ref_val[5] = 42;
  ref_key[6] = 4; ref_val[6] = 13;

  Vector result_key(7), result_val(7);

  hipStream_t s;
  hipStreamCreate(&s);

  thrust::pair<Iterator,Iterator> ends =
    thrust::merge_by_key(thrust::cuda::par(s),
                         a_key.begin(), a_key.end(),
                         b_key.begin(), b_key.end(),
                         a_val.begin(), b_val.begin(),
                         result_key.begin(),
                         result_val.begin());

  hipStreamSynchronize(s);

  ASSERT_EQUAL_QUIET(result_key.end(), ends.first);
  ASSERT_EQUAL_QUIET(result_val.end(), ends.second);
  ASSERT_EQUAL(ref_key, result_key);
  ASSERT_EQUAL(ref_val, result_val);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestMergeByKeyCudaStreams);

