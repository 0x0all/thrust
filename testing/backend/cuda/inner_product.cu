#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/inner_product.h>
#include <thrust/execution_policy.h>


template<typename Iterator1, typename Iterator2, typename T, typename Iterator3>
__global__
void inner_product_kernel(Iterator1 first1, Iterator1 last1, Iterator2 first2, T init, Iterator3 result)
{
  *result = thrust::inner_product(thrust::seq, first1, last1, first2, init);
}


template<typename T>
struct TestInnerProductDeviceSeq
{
  void operator()(const size_t n)
  {
    thrust::host_vector<T> h_v1 = unittest::random_integers<T>(n);
    thrust::host_vector<T> h_v2 = unittest::random_integers<T>(n);
    
    thrust::device_vector<T> d_v1 = h_v1;
    thrust::device_vector<T> d_v2 = h_v2;

    thrust::device_vector<T> result(1);
    
    T init = 13;
    
    T expected = thrust::inner_product(h_v1.begin(), h_v1.end(), h_v2.begin(), init);
    inner_product_kernel<<<1,1>>>(d_v1.begin(), d_v1.end(), d_v2.begin(), init, result.begin());
    
    ASSERT_EQUAL(expected, result[0]);
  }
};
VariableUnitTest<TestInnerProductDeviceSeq, IntegralTypes> TestInnerProductDeviceSeqInstance;


void TestInnerProductCudaStreams()
{
  thrust::device_vector<int> v1(3);
  thrust::device_vector<int> v2(3);
  v1[0] =  1; v1[1] = -2; v1[2] =  3;
  v2[0] = -4; v2[1] =  5; v2[2] =  6;

  hipStream_t s;
  hipStreamCreate(&s);
  
  int init = 3;
  int result = thrust::inner_product(thrust::cuda::par(s), v1.begin(), v1.end(), v2.begin(), init);
  ASSERT_EQUAL(result, 7);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestInnerProductCudaStreams);

