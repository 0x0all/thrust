#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/reverse.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator>
__global__
void reverse_kernel(ExecutionPolicy exec, Iterator first, Iterator last)
{
  thrust::reverse(exec, first, last);
}


template<typename T, typename ExecutionPolicy>
void TestReverseDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T> h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::reverse(h_data.begin(), h_data.end());
  reverse_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end());
  
  ASSERT_EQUAL(h_data, d_data);
};

template<typename T>
void TestReverseDeviceSeq(const size_t n)
{
  TestReverseDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestReverseDeviceSeq);

template<typename T>
void TestReverseDeviceDevice(const size_t n)
{
  TestReverseDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestReverseDeviceDevice);


template<typename ExecutionPolicy, typename Iterator1, typename Iterator2>
__global__
void reverse_copy_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result)
{
  thrust::reverse_copy(exec, first, last, result);
}


template<typename T, typename ExecutionPolicy>
void TestReverseCopyDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T> h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;

  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);

  thrust::reverse_copy(h_data.begin(), h_data.end(), h_result.begin());
  reverse_copy_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
};

template<typename T>
void TestReverseCopyDeviceSeq(const size_t n)
{
  TestReverseCopyDevice<T>(thrust::seq, n);
}
DECLARE_VARIABLE_UNITTEST(TestReverseCopyDeviceSeq);

template<typename T>
void TestReverseCopyDeviceDevice(const size_t n)
{
  TestReverseCopyDevice<T>(thrust::device, n);
}
DECLARE_VARIABLE_UNITTEST(TestReverseCopyDeviceDevice);

