#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename Iterator, typename Function, typename Iterator2>
__global__
void all_of_kernel(Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::all_of(thrust::seq, first, last, f);
}


void TestAllOfDeviceSeq()
{
  typedef int T;
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  all_of_kernel<<<1,1>>>(v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
  
  v[1] = 0;
  
  all_of_kernel<<<1,1>>>(v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
  
  all_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  all_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  all_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  all_of_kernel<<<1,1>>>(v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
}
DECLARE_UNITTEST(TestAllOfDeviceSeq);


void TestAllOfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;
  
  Vector v(3, 1);

  hipStream_t s;
  hipStreamCreate(&s);
  
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par(s), v.begin(), v.end(), thrust::identity<T>()), true);
  
  v[1] = 0;
  
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par(s), v.begin(), v.end(), thrust::identity<T>()), false);
  
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 0, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 1, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 2, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::all_of(thrust::cuda::par(s), v.begin() + 1, v.begin() + 2, thrust::identity<T>()), false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestAllOfCudaStreams);


template<typename Iterator, typename Function, typename Iterator2>
__global__
void any_of_kernel(Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::any_of(thrust::seq, first, last, f);
}


void TestAnyOfDeviceSeq()
{
  typedef int T;
  
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  any_of_kernel<<<1,1>>>(v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
  
  v[1] = 0;
  
  any_of_kernel<<<1,1>>>(v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
  
  any_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  any_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  any_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  any_of_kernel<<<1,1>>>(v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
}
DECLARE_UNITTEST(TestAnyOfDeviceSeq);


void TestAnyOfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;

  Vector v(3, 1);

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par(s), v.begin(), v.end(), thrust::identity<T>()), true);

  v[1] = 0;
  
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par(s), v.begin(), v.end(), thrust::identity<T>()), true);

  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 0, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 1, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 2, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::any_of(thrust::cuda::par(s), v.begin() + 1, v.begin() + 2, thrust::identity<T>()), false);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestAnyOfCudaStreams);


template<typename Iterator, typename Function, typename Iterator2>
__global__
void none_of_kernel(Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::none_of(thrust::seq, first, last, f);
}


void TestNoneOfDeviceSeq()
{
  typedef int T;
  
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  none_of_kernel<<<1,1>>>(v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
  
  v[1] = 0;
  
  none_of_kernel<<<1,1>>>(v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
  
  none_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  none_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  none_of_kernel<<<1,1>>>(v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  none_of_kernel<<<1,1>>>(v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
}
DECLARE_UNITTEST(TestNoneOfDeviceSeq);


void TestNoneOfCudaStreams()
{
  typedef thrust::device_vector<int> Vector;
  typedef typename Vector::value_type T;

  Vector v(3, 1);

  hipStream_t s;
  hipStreamCreate(&s);

  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par(s), v.begin(), v.end(), thrust::identity<T>()), false);

  v[1] = 0;
  
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par(s), v.begin(), v.end(), thrust::identity<T>()), false);

  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 0, thrust::identity<T>()), true);
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 1, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par(s), v.begin() + 0, v.begin() + 2, thrust::identity<T>()), false);
  ASSERT_EQUAL(thrust::none_of(thrust::cuda::par(s), v.begin() + 1, v.begin() + 2, thrust::identity<T>()), true);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestNoneOfCudaStreams);

