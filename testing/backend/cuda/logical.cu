#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/logical.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename Function, typename Iterator2>
__global__
void all_of_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::all_of(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestAllOfDevice(ExecutionPolicy exec)
{
  typedef int T;
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  all_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
  
  v[1] = 0;
  
  all_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
  
  all_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  all_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  all_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  all_of_kernel<<<1,1>>>(exec, v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
}


void TestAllOfDeviceSeq()
{
  TestAllOfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestAllOfDeviceSeq);


void TestAllOfDeviceDevice()
{
  TestAllOfDevice(thrust::device);
}
DECLARE_UNITTEST(TestAllOfDeviceDevice);


template<typename ExecutionPolicy, typename Iterator, typename Function, typename Iterator2>
__global__
void any_of_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::any_of(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestAnyOfDevice(ExecutionPolicy exec)
{
  typedef int T;
  
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  any_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
  
  v[1] = 0;
  
  any_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
  
  any_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  any_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  any_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  any_of_kernel<<<1,1>>>(exec, v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
}


void TestAnyOfDeviceSeq()
{
  TestAnyOfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestAnyOfDeviceSeq);


void TestAnyOfDeviceDevice()
{
  TestAnyOfDevice(thrust::device);
}
DECLARE_UNITTEST(TestAnyOfDeviceDevice);


template<typename ExecutionPolicy, typename Iterator, typename Function, typename Iterator2>
__global__
void none_of_kernel(ExecutionPolicy exec, Iterator first, Iterator last, Function f, Iterator2 result)
{
  *result = thrust::none_of(exec, first, last, f);
}


template<typename ExecutionPolicy>
void TestNoneOfDevice(ExecutionPolicy exec)
{
  typedef int T;
  
  thrust::device_vector<T> v(3, 1);
  thrust::device_vector<bool> result(1);
  
  none_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
  
  v[1] = 0;
  
  none_of_kernel<<<1,1>>>(exec, v.begin(), v.end(), thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);
  
  none_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 0, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);

  none_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 1, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  none_of_kernel<<<1,1>>>(exec, v.begin() + 0, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(false, result[0]);

  none_of_kernel<<<1,1>>>(exec, v.begin() + 1, v.begin() + 2, thrust::identity<T>(), result.begin());
  ASSERT_EQUAL(true, result[0]);
}


void TestNoneOfDeviceSeq()
{
  TestNoneOfDevice(thrust::seq);
}
DECLARE_UNITTEST(TestNoneOfDeviceSeq);


void TestNoneOfDeviceDevice()
{
  TestNoneOfDevice(thrust::device);
}
DECLARE_UNITTEST(TestNoneOfDeviceDevice);

